#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "light/envmap.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include "light/areaLight.h"
#include <vector>

using namespace optix;


rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );
rtDeclareVariable(int, max_depth, , );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable( float, uvScale, , ); 

// Materials
rtDeclareVariable( float3, albedo, , );
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );
rtDeclareVariable( float3, specular, , );
rtTextureSampler<float4, 2> specularMap;
rtDeclareVariable( int, isSpecularTexture, , );
rtDeclareVariable( float, glossy, , );
rtTextureSampler<float4, 2> glossyMap;
rtDeclareVariable( int, isGlossyTexture, , );
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable(int, isNormalTexture, , );
rtDeclareVariable(float, F0, , );

// Area Light Buffer
rtDeclareVariable(int, isAreaLight, , );

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtDeclareVariable(float, infiniteFar, , );

// Point lighting 
rtDeclareVariable(int, isPointLight, , );
rtDeclareVariable(int, pointLightNum, , );
rtBuffer<Point> pointLights;


// Geometry Group
rtDeclareVariable( rtObject, top_object, , );

rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float&)>, 
        sampleEnvironmapLight, , );
rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float3&, float&)>, 
        sampleAreaLight, , );


// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float LambertianPdf(const float3& L, const float3& N)
{
    float NoL = fmaxf(dot(N, L), 1e-14);
    float pdf = NoL / M_PIf;
    return fmaxf(pdf, 1e-14f);
}
RT_CALLABLE_PROGRAM float SpecularPdf(const float3& L, const float3& N, const float3& R, 
        float glossyValue)
{ 
    float RoL = dot(R, L);
    if(RoL < 1e-14) RoL = 0;
    float pdf = (glossyValue + 2) / (2*M_PIf) * pow(RoL, fmaxf(glossyValue, 1e-14) );
    return fmaxf(pdf, 1e-14);
}
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& N, const float3& R, const float3& albedoValue, const float3& specularValue, float glossyValue)
{
    float pdfLambertian = LambertianPdf(L, N);
    float pdfSpecular = SpecularPdf(L, N, R, glossyValue);
    float albedoStr = length(albedoValue );
    float specularStr = length(specularValue );
    float pdf = (albedoStr * pdfLambertian + specularStr * pdfLambertian) / fmaxf(albedoStr + specularStr, 1e-14);
    return fmaxf(pdf, 1e-14);
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& albedoValue, const float3& specularValue, const float3& N, const float glossyValue, 
        const float3& L, const float3& R, const float3& radiance)
{
    float NoL = fmaxf(dot(N, L), 1e-14);

    float RoL = dot(R, L);
    if(RoL < 1e-14) RoL = 0;

    float3 lambertianTerm = albedoValue / M_PIf;
    float3 specularTerm = specularValue / (2*M_PIf) * (glossyValue + 2) * pow(RoL, fmaxf(glossyValue, 1e-14) );
    return (lambertianTerm + specularTerm) * radiance * NoL;
}

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        const float3& albedoValue, const float3& specularValue, const float3& N, const float glossyValue, const float3& R, 
        optix::Onb onb, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    const float z1 = rnd( seed );
    const float z2 = rnd( seed );
    const float z = rnd( seed );
    
    float albedoStr = length(albedoValue );
    float specularStr = length(specularValue );

    float3 L;
    if(z <= albedoStr / fmaxf(albedoStr + specularStr, 1e-14) || (albedoStr + specularStr) < 1e-14 ){
        cosine_sample_hemisphere(z1, z2, L);
        onb.inverse_transform(L);
        attenuation = attenuation * albedoValue * (albedoStr + specularStr) / fmaxf(albedoStr, 1e-14);
    }
    else{
        float z1_1_nP1 = pow(z1, 1 / (glossyValue +1) );
        float z1_2_nP1 = z1_1_nP1 * z1_1_nP1;
        L = make_float3(
                sqrt(1 - z1_2_nP1) * cos(2 * M_PIf * z2), 
                sqrt(1 - z1_2_nP1) * sin(2 * M_PIf * z2),
                z1_1_nP1
                );
        optix::Onb ronb(R);
        ronb.inverse_transform(L);
        float NoL = fmaxf(dot(N, L), 1e-14);
        attenuation = attenuation * specularValue * NoL * (albedoStr + specularStr) / fmaxf(specularStr, 1e-14); 
    }
    direction = L;
    pdfSolid = pdf(L, N, R, albedoValue, specularValue, glossyValue);
    return;
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
 
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x * uvScale, texcoord.y * uvScale ) );
    }

    float3 specularValue;
    if(isSpecularTexture == 0){
        specularValue = specular;
    }
    else{
        specularValue = make_float3(tex2D(specularMap, texcoord.x * uvScale, texcoord.y * uvScale) );
    }


    float3 colorSum = fmaxf(albedoValue + specularValue, make_float3(1e-14f) );
    float colorMax= colorSum.x;
    if(colorMax < colorSum.y) colorMax = colorSum.y;
    if(colorMax < colorSum.z) colorMax = colorSum.z;
    colorMax = fmaxf(colorMax, 1e-14);

    if(colorMax > 1){
        specularValue = specularValue / colorMax;
        albedoValue = albedoValue / colorMax;
    }

    float glossyValue = (isGlossyTexture == 0) ? glossy :
        tex2D(glossyMap, texcoord.x, texcoord.y).x;
    
    float3 V = normalize(-ray.direction );    
    if(dot(ffnormal, V) < 0)
        ffnormal = -ffnormal;
    
    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal;
    }
    N = normalize(N );
    optix::Onb onb(N );
    
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    float3 R = 2 * dot(V, N) * N - V;

    // Connect to the area Light
    {
        if(isAreaLight == 1){
            float3 position, radiance, normal;
            float pdfAreaLight;
            sampleAreaLight(prd_radiance.seed, radiance, position, normal, pdfAreaLight);
   
            float Dist = length(position - hitPoint);
            float3 L = normalize(position - hitPoint);

            if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(N, V), 0.0) > 0 ){
                float cosPhi = dot(L, normal);
                cosPhi = (cosPhi < 0) ? -cosPhi : cosPhi;

                Ray shadowRay = make_Ray(hitPoint, L, 1, scene_epsilon, Dist - scene_epsilon);
                PerRayData_shadow prd_shadow; 
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance) * cosPhi / Dist / Dist;
                    if(prd_radiance.depth == (max_depth - 1) ){
                    }
                    else{
                        float pdfSolidBRDF = pdf(L, N, R, albedoValue, specularValue, glossyValue);
                        float pdfAreaBRDF = pdfSolidBRDF * cosPhi / Dist / Dist;

                        float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
                        float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;

                        prd_radiance.radiance += intensity * pdfAreaLight / (pdfAreaBRDF2 + pdfAreaLight2) * prd_radiance.attenuation;
                    }
                }
            }
        }
    }
    
    // Connect to point light 
    {
        if(isPointLight == 1){
            // Connect to every point light 
            for(int i = 0; i < pointLightNum; i++){
                float3 position = pointLights[i].position;
                float3 radiance = pointLights[i].intensity;
                float3 L = normalize(position - hitPoint);
                float Dist = length(position - hitPoint);

                if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(N, V), 0.0) > 0 ){
                    Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                    PerRayData_shadow prd_shadow; 
                    prd_shadow.inShadow = false;
                    rtTrace(top_object, shadowRay, prd_shadow);
                    if(prd_shadow.inShadow == false && prd_radiance.depth != (max_depth - 1 ) )
                    {
                        float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance) / Dist/ Dist;
                        prd_radiance.radiance += intensity * prd_radiance.attenuation;
                    }
                }
            }
        }
    }

    // Connect to the environmental map 
    { 
        if(isEnvmap == 1){
            float3 L, radiance;
            float pdfSolidEnv;
            sampleEnvironmapLight(prd_radiance.seed, radiance, L, pdfSolidEnv);

            if( fmaxf(dot(L, N), 0.0) * fmaxf(dot(V, N), 0.0) > 0){
                Ray shadowRay = make_Ray(hitPoint + 0.1 * scene_epsilon * L, L, 1, scene_epsilon, infiniteFar);
                PerRayData_shadow prd_shadow;
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, specularValue, N, glossyValue, L, R, radiance);
                    if(prd_radiance.depth == (max_depth - 1) ){
                    }
                    else{
                        float pdfSolidBRDF = pdf(L, N, R, albedoValue, specularValue, glossyValue);
                        float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;
                        float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv;
                        prd_radiance.radiance += intensity * pdfSolidEnv / (pdfSolidEnv2 + pdfSolidBRDF2) * prd_radiance.attenuation; 
                    }
                }
            }
        }
    }

    // Sammple the new ray 
    sample(prd_radiance.seed, 
        albedoValue, specularValue, N, glossyValue, R, 
        onb, 
        prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf);

}    

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

