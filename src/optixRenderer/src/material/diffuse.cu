#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "light/envmap.h"
#include "light/areaLight.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include <vector>

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );
rtDeclareVariable(int, max_depth, , );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable( float, uvScale, , ); 

// Diffuse albedo
rtDeclareVariable( float3, albedo, , );
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable( int, isNormalTexture, , );


// The area Light Buffer
rtDeclareVariable(int, isAreaLight, , );

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtDeclareVariable(float, infiniteFar, , );

// Point lighting 
rtDeclareVariable(int, isPointLight, , );
rtDeclareVariable(int, pointLightNum, , );
rtBuffer<Point> pointLights;

// Geometry Group
rtDeclareVariable( rtObject, top_object, , );

rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float&)>, 
        sampleEnvironmapLight, , );
rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float3&, float&)>, 
        sampleAreaLight, , );


// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& V, const float3& N)
{
    float NoL = fmaxf(dot(N, L), 0);
    float pdf = NoL / M_PIf; 
    return fmaxf(pdf, 1e-14);
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& albedoValue, const float3& N, const float3& V, const float3& L, const float3& radiance)
{   
    float NoL = fmaxf(dot(N, L), 1e-14);
    float3 intensity = albedoValue / M_PIf * NoL * radiance; 
    return intensity;
}

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        const float3& albedoValue, const float3& N, const float3& V, 
        optix::Onb& onb, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    const float z1 = rnd( seed );
    const float z2 = rnd( seed );

    float3 L;
    cosine_sample_hemisphere(z1, z2, L);
    onb.inverse_transform(L);
    direction = L;
    attenuation = attenuation * albedoValue;
    pdfSolid = pdf(L, V, N);
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x * uvScale, texcoord.y * uvScale ) );
    }
   
    float3 V = normalize(-ray.direction );
    if(dot(ffnormal, V) < 0)
        ffnormal = -ffnormal;
    
    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x * uvScale, texcoord.y * uvScale ) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal;
    }
    N = normalize(N );
    optix::Onb onb(N );

    
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    // Connect to the area Light
    {
        if(isAreaLight == 1){
            float3 position, radiance, normal;
            float pdfAreaLight;
            sampleAreaLight(prd_radiance.seed, radiance, position, normal, pdfAreaLight);
   
            float Dist = length(position - hitPoint);
            float3 L = normalize(position - hitPoint);

            if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(V, N), 0.0) > 0 ){
                float cosPhi = dot(L, normal);
                cosPhi = (cosPhi < 0) ? -cosPhi : cosPhi;

                Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                PerRayData_shadow prd_shadow; 
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, V, L, radiance) * cosPhi / Dist / Dist;

                    if(prd_radiance.depth == (max_depth-1) ){
                    }
                    else{
                        float pdfSolidBRDF = pdf(L, V, N);
                        float pdfAreaBRDF = pdfSolidBRDF * cosPhi / Dist / Dist;

                        float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
                        float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;

                        prd_radiance.radiance += intensity * pdfAreaLight / fmaxf(pdfAreaBRDF2 + pdfAreaLight2, 1e-14) * prd_radiance.attenuation;
                    }
                    
                }
            }
        }
    }

    // Connect to point light 
    {
        if(isPointLight == 1){
            // Connect to every point light 
            for(int i = 0; i < pointLightNum; i++){
                float3 position = pointLights[i].position;
                float3 radiance = pointLights[i].intensity;
                float3 L = normalize(position - hitPoint);
                float Dist = length(position - hitPoint);
                
                if(fmaxf(dot(N, L), 0.0) * fmaxf(dot(N, V), 0.0) > 0 ){
                    Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                    PerRayData_shadow prd_shadow; 
                    prd_shadow.inShadow = false;
                    rtTrace(top_object, shadowRay, prd_shadow);
                    if(prd_shadow.inShadow == false && prd_radiance.depth != (max_depth-1) ){
                        float3 intensity = evaluate(albedoValue, N, V, L, radiance) / Dist / Dist;
                        prd_radiance.radiance += intensity * prd_radiance.attenuation;
                    }
                }
            }
        }
    }

    // Connect to the environmental map 
    { 
        if(isEnvmap == 1){
            float3 L, radiance;
            float pdfSolidEnv;
            sampleEnvironmapLight(prd_radiance.seed, radiance, L, pdfSolidEnv);

            if( fmaxf(dot(L, N), 0.0) * fmaxf(dot(V, N), 0.0)  > 0 ){
                Ray shadowRay = make_Ray(hitPoint + 0.1*scene_epsilon*L, L, 1, scene_epsilon, infiniteFar);
                PerRayData_shadow prd_shadow;
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, V, L, radiance);
                    if(prd_radiance.depth == (max_depth-1) ){
                    }
                    else{
                        float pdfSolidBRDF = pdf(L, V, N);
                        float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;
                        float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv;
                        prd_radiance.radiance += intensity * pdfSolidEnv /
                            fmaxf( (pdfSolidEnv2 + pdfSolidBRDF2), 1e-14) * prd_radiance.attenuation; 
                    }
                }
            }
        }
    }

    // Finish updating the ray
    sample(prd_radiance.seed,
            albedoValue, N, V,
            onb,
            prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf);
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

