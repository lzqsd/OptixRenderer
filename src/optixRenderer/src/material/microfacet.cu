#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "light/envmap.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include "light/areaLight.h"
#include <vector>

using namespace optix;


rtDeclareVariable( float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );
rtDeclareVariable(int, max_depth, , );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

rtDeclareVariable( float, uvScale, , ); 

// Materials
rtDeclareVariable( float3, albedo, , );
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );
rtDeclareVariable( float, rough, , );
rtTextureSampler<float4, 2> roughMap;
rtDeclareVariable( int, isRoughTexture, , );
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable(int, isNormalTexture, , );
rtDeclareVariable(float, F0, , );
rtDeclareVariable( float, metallic, , );
rtDeclareVariable( int, isMetallicTexture, ,  );
rtTextureSampler<float4, 2> metallicMap;

// Area Light Buffer
rtDeclareVariable(int, isAreaLight, , );

// Environmental Lighting 
rtDeclareVariable(int, isEnvmap, , );
rtDeclareVariable(float, infiniteFar, , );

// Point lighting 
rtDeclareVariable(int, isPointLight, , );
rtDeclareVariable(int, pointLightNum, , );
rtBuffer<Point> pointLights;


// Geometry Group
rtDeclareVariable( rtObject, top_object, , );

rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float&)>, 
        sampleEnvironmapLight, , );
rtDeclareVariable(
        rtCallableProgramX<void(unsigned int&, float3&, float3&, float3&, float&)>, 
        sampleAreaLight, , );


// Computing the pdfSolidAngle of BRDF giving a direction 
RT_CALLABLE_PROGRAM float LambertianPdf(const float3& L, const float3& N)
{
    float NoL = fmaxf(dot(N, L), 0);
    float pdf = NoL / M_PIf;
    return fmaxf(pdf, 1e-14f);
}
RT_CALLABLE_PROGRAM float SpecularPdf(const float3& L, const float3& V, const float3& N, float R)
{
    float a2 = R * R * R * R;
    float3 H = normalize( (L+V) / 2.0 );
    float NoH = fmaxf(dot(N, H), 0);
    float VoH = fmaxf(dot(V, H), 0);
    float pdf = (a2 * NoH) / fmaxf( (4 * M_PIf * (1 + (a2-1) * NoH)
            *(1 + (a2-1) * NoH) * VoH ), 1e-14f);
    return fmaxf(pdf, 1e-14f);
}
RT_CALLABLE_PROGRAM float pdf(const float3& L, const float3& V, const float3& N, float R)
{
    float pdfLambertian = LambertianPdf(L, N);
    float pdfSpecular = SpecularPdf(L, V, N, R);
    return pdfLambertian * 0.5 + pdfSpecular * 0.5;
}

RT_CALLABLE_PROGRAM float3 evaluate(const float3& albedoValue, const float3& N, const float rough, const float3& fresnel, 
        const float3& V, const float3& L, const float3& radiance)
{
    float alpha = rough * rough;
    float k = (alpha + 2 * rough + 1) / 8.0;
    float alpha2 = alpha * alpha;
    
    float3 H = normalize((L + V) / 2.0f );
    float NoL = fmaxf(dot(N, L), 0);
    float NoV = fmaxf(dot(N, V), 0);
    float NoH = fmaxf(dot(N, H), 0);
    float VoH = fmaxf(dot(V, H), 0);

    float FMi = (-5.55473 * VoH - 6.98316) * VoH;
    float3 frac0 = fresnel + (1 - fresnel) * pow(2.0f, FMi);
    float3 frac = frac0 * alpha2;
    float nom0 = NoH * NoH * (alpha2 - 1) + 1;
    float nom1 = NoV * (1 - k) + k;
    float nom2 = NoL * (1 - k) + k;
    float nom = fmaxf(4 * M_PIf * nom0 * nom0 * nom1 * nom2, 1e-14);
    float3 spec = frac / nom;
         
    float3 intensity = (albedoValue / M_PIf + spec) * NoL * radiance; 
    return intensity;
}

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        const float3& albedoValue, const float3& N, const float rough, const float3& fresnel, const float3& V,  
        const float3& ffnormal, 
        optix::Onb onb, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
    const float z1 = rnd( seed );
    const float z2 = rnd( seed );
    const float z = rnd( seed );
    
    float alpha = rough * rough;
    float k = (alpha + 2 * rough + 1) / 8.0;
    float alpha2 = alpha * alpha;
    
    float3 L;
    if(z < 0.5 ){
        cosine_sample_hemisphere(z1, z2, L);
        onb.inverse_transform(L);
        direction = L;
        attenuation =  2 * attenuation * albedoValue;
    }
    else{
        // Compute the half angle 
        float phi = 2 * M_PIf * z1;
        float cosTheta = sqrt( (1 - z2) / (1 + (alpha2 - 1) * z2) );
        float sinTheta = sqrt( 1 - cosTheta * cosTheta);

        float3 H = make_float3(
                sinTheta * cos(phi),
                sinTheta * sin(phi),
                cosTheta );
        onb.inverse_transform(H);
        L = 2 * dot(V, H) * H - V;
        direction = L;

        float NoV = fmaxf(dot(N, V), 0.0);
        float NoL = fmaxf(dot(N, L), 0.0);
        float NoH = fmaxf(dot(N, H), 0.0);
        float VoH = fmaxf(dot(V, H), 0.0);

        if( dot(ffnormal, L) >= 0.05 ){
            float G1 = NoV / (NoV * (1-k) + k);
            float G2 = NoL / (NoL * (1-k) + k);
            float FMi = (-5.55473 * VoH - 6.98316) * VoH;
            float3 F = fresnel + (1 - fresnel) * pow(2.0f, FMi);
            float3 reflec = F * G1 * G2 * VoH / fmaxf(NoH * NoV, 1e-14);

            attenuation = 2 * attenuation * reflec;
        }
        else{
            attenuation = make_float3(0.0f);
        }
    }
    pdfSolid = pdf(L, V, N, rough);
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
 
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x * uvScale, texcoord.y * uvScale) );
    }

    float roughValue = (isRoughTexture == 0) ? rough :
        tex2D(roughMap, texcoord.x * uvScale, texcoord.y * uvScale).x;

    float metallicValue = (isMetallicTexture == 0) ? metallic :
        tex2D(metallicMap, texcoord.x * uvScale, texcoord.y * uvScale).x;

    float3 fresnel = F0 * (1 - metallicValue) + metallicValue * albedoValue;
    albedoValue = (1 - metallicValue) * albedoValue;
    
    float3 V = normalize(-ray.direction );    
    if(dot(ffnormal, V) < 0)
        ffnormal = -ffnormal;
    
    float3 N;
    if( isNormalTexture == 0){
        N = ffnormal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x * uvScale, texcoord.y * uvScale) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * ffnormal;
    } 
    N = normalize(N );
    optix::Onb onb(N );
 
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    // Connect to the area Light
    {
        if(isAreaLight == 1){
            float3 position, radiance, normal;
            float pdfAreaLight;
            sampleAreaLight(prd_radiance.seed, radiance, position, normal, pdfAreaLight);
   
            float Dist = length(position - hitPoint);
            float3 L = normalize(position - hitPoint);

            if(fmaxf(dot(ffnormal, L), 0.0f) * fmaxf(dot(ffnormal, V), 0.0f) > 0.0025 ){
                float cosPhi = dot(L, normal);
                cosPhi = (cosPhi < 0) ? -cosPhi : cosPhi;

                Ray shadowRay = make_Ray(hitPoint, L, 1, scene_epsilon, Dist - scene_epsilon);
                PerRayData_shadow prd_shadow; 
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance) * cosPhi / Dist / Dist;
                    
                    if(prd_radiance.depth == (max_depth - 1) ){
                    }
                    else{
                        float pdfAreaLight2 = pdfAreaLight * pdfAreaLight;
                        float pdfSolidBRDF = pdf(L, V, N, roughValue);
                        float pdfAreaBRDF = pdfSolidBRDF * cosPhi / Dist / Dist;
                        float pdfAreaBRDF2 = pdfAreaBRDF * pdfAreaBRDF;
                        
                        float3 radianceInc = intensity * pdfAreaLight / 
                            fmaxf(pdfAreaBRDF2 + pdfAreaLight2, 1e-14) * prd_radiance.attenuation;            
                        prd_radiance.radiance += radianceInc;
                    }
                }
            }
        }   
    }

    
    // Connect to point light 
    {
        if(isPointLight == 1){
            // Connect to every point light 
            for(int i = 0; i < pointLightNum; i++){
                float3 position = pointLights[i].position;
                float3 radiance = pointLights[i].intensity;
                float3 L = normalize(position - hitPoint);
                float Dist = length(position - hitPoint);

                if( fmaxf(dot(ffnormal, L), 0.0f) * fmaxf(dot(ffnormal, V), 0.0f) > 0.0025){
                    Ray shadowRay = make_Ray(hitPoint + 0.1 * L * scene_epsilon, L, 1, scene_epsilon, Dist - scene_epsilon);
                    PerRayData_shadow prd_shadow; 
                    prd_shadow.inShadow = false;
                    rtTrace(top_object, shadowRay, prd_shadow);
                    if(prd_shadow.inShadow == false && prd_radiance.depth != (max_depth - 1) ){
                        float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance) / Dist/ Dist; 

                        prd_radiance.radiance += intensity * prd_radiance.attenuation;
                    }
                }
            }
        }
    }

    // Connect to the environmental map 
    { 
        if(isEnvmap == 1){
            float3 L, radiance;
            float pdfSolidEnv;
            sampleEnvironmapLight(prd_radiance.seed, radiance, L, pdfSolidEnv);

            if( fmaxf(dot(L, ffnormal), 0.0f) * fmaxf(dot(V, ffnormal ), 0.0f) > 0.0025){
                Ray shadowRay = make_Ray(hitPoint + 0.1 * scene_epsilon * L, L, 1, scene_epsilon, infiniteFar);
                PerRayData_shadow prd_shadow;
                prd_shadow.inShadow = false;
                rtTrace(top_object, shadowRay, prd_shadow);
                if(prd_shadow.inShadow == false)
                {
                    float3 intensity = evaluate(albedoValue, N, roughValue, fresnel, V, L, radiance);
                    if(prd_radiance.depth == (max_depth - 1) ){
                    }
                    else{
                        float pdfSolidBRDF = pdf(L, V, N, roughValue);
                        float pdfSolidBRDF2 = pdfSolidBRDF * pdfSolidBRDF;
                        float pdfSolidEnv2 = pdfSolidEnv * pdfSolidEnv; 

                        float3 radianceInc = intensity * pdfSolidEnv / 
                            fmaxf(pdfSolidEnv2 + pdfSolidBRDF2, 1e-14) * prd_radiance.attenuation;  
                        prd_radiance.radiance += radianceInc; 
                    }
                }
            }
        }
    }


    // Sammple the new ray 
    sample(prd_radiance.seed, 
        albedoValue, N, fmaxf(roughValue, 0.02), fresnel, V, 
        ffnormal, onb, 
        prd_radiance.attenuation, prd_radiance.direction, prd_radiance.pdf );
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

