#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include <vector>

using namespace optix;

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

// Diffuse albedo
rtDeclareVariable( float3, albedo, , ); 
rtDeclareVariable( float, uvScale, , ); 
rtTextureSampler<float4, 2> albedoMap;
rtDeclareVariable( int, isAlbedoTexture, , );

RT_PROGRAM void closest_hit_radiance()
{
    float3 albedoValue;
    if(isAlbedoTexture == 0){
        albedoValue = albedo;
    }
    else{
        albedoValue = make_float3(tex2D(albedoMap, texcoord.x * uvScale, texcoord.y * uvScale) );
    }
    prd_radiance.radiance = albedoValue;
    prd_radiance.done = true;
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

